#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define SHARED_MEMORY_MAX_SIZE 1024
#define UINT_SIZE 32

#define ERROR_HANDLING(call) { \
    hipError_t error = call; \
    if (error != hipSuccess) { \
        fprintf(stderr, "ERROR: file '%s' in line %d: %s.\n", __FILE__, __LINE__, hipGetErrorString(error)); \
        exit(1); \
    } \
}

// Функция для вывода массива (для отладки)
void print_array(uint *a, int n) {
    for (int i = 0; i < n; i++)
        printf("%u ", a[i]);
    printf("\n");
}

// Устройство: Сканирование (префиксная сумма) для одного блока
__global__
void scan_block(int *input, int *output, int n) {
    extern __shared__ int temp[];

    int th_id = threadIdx.x;
    int offset = 1;

    int left_idx = 2 * th_id;
    int right_idx = 2 * th_id + 1;

    if (left_idx < n) temp[left_idx] = input[left_idx];
    else temp[left_idx] = 0;

    if (right_idx < n) temp[right_idx] = input[right_idx];
    else temp[right_idx] = 0;

    // Построение верхнего уровня дерева
    for (int d = n >> 1; d > 0; d >>= 1) {
        __syncthreads();
        if (th_id < d) {
            int ai = offset * (2 * th_id + 1) - 1;
            int bi = offset * (2 * th_id + 2) - 1;
            temp[bi] += temp[ai];
        }
        offset <<= 1;
    }

    // Установка корня в 0
    if (th_id == 0)
        temp[n - 1] = 0;

    // Префиксная сумма
    for (int d = 1; d < n; d <<= 1) {
        offset >>= 1;
        __syncthreads();
        if (th_id < d) {
            int ai = offset * (2 * th_id + 1) - 1;
            int bi = offset * (2 * th_id + 2) - 1;
            int t = temp[ai];
            temp[ai] = temp[bi];
            temp[bi] += t;
        }
    }
    __syncthreads();

    if (left_idx < n) output[left_idx] = temp[left_idx];
    if (right_idx < n) output[right_idx] = temp[right_idx];
}

// Устройство: Получение разряда числа
__global__
void extract_bit(uint *input, int *bit_array, int digit, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        bit_array[idx] = (input[idx] >> digit) & 1;
    }
}

// Устройство: Перестановка элементов на основе битов
__global__
void rearrange(uint *output, uint *input, int *prefix_sums, int *bits, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        int pos = (bits[idx] == 0) ? idx - prefix_sums[idx] : prefix_sums[idx] + (n - prefix_sums[n - 1]);
        if (pos >= 0 && pos < n) {
            output[pos] = input[idx];
        }
    }
}

// Хост: Полная поразрядная сортировка
void radix_sort(uint *array, int n) {
    uint *d_input, *d_output;
    int *d_bits, *d_prefix_sums;

    ERROR_HANDLING(hipMalloc(&d_input, n * sizeof(uint)));
    ERROR_HANDLING(hipMalloc(&d_output, n * sizeof(uint)));
    ERROR_HANDLING(hipMalloc(&d_bits, n * sizeof(int)));
    ERROR_HANDLING(hipMalloc(&d_prefix_sums, n * sizeof(int)));

    ERROR_HANDLING(hipMemcpy(d_input, array, n * sizeof(uint), hipMemcpyHostToDevice));

    for (int digit = 0; digit < UINT_SIZE; digit++) {
        // Извлечение разряда
        extract_bit<<<(n + 255) / 256, 256>>>(d_input, d_bits, digit, n);

        // Префиксная сумма
        int shared_memory_size = (n > SHARED_MEMORY_MAX_SIZE) ? SHARED_MEMORY_MAX_SIZE : n;
        scan_block<<<1, shared_memory_size / 2, shared_memory_size * sizeof(int)>>>(d_bits, d_prefix_sums, n);

        // Перестановка элементов
        rearrange<<<(n + 255) / 256, 256>>>(d_output, d_input, d_prefix_sums, d_bits, n);

        // Обновление массивов
        uint *temp = d_input;
        d_input = d_output;
        d_output = temp;
    }

    ERROR_HANDLING(hipMemcpy(array, d_input, n * sizeof(uint), hipMemcpyDeviceToHost));

    hipFree(d_input);
    hipFree(d_output);
    hipFree(d_bits);
    hipFree(d_prefix_sums);
}

// Основная функция
int main() {
    int n;
    fread(&n, sizeof(int), 1, stdin);

    uint *array = (uint *)malloc(n * sizeof(uint));
    fread(array, sizeof(uint), n, stdin);

    radix_sort(array, n);

    fwrite(array, sizeof(uint), n, stdout);

    free(array);
    return 0;
}